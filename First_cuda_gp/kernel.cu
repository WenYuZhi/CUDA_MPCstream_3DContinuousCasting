#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "book.h"
#include "gridcheck.h"

# define Section 12  // number of cooling sections
# define CoolSection 8
# define MoldSection 4

float ccml[Section + 1] = { 0.0,0.2,0.4,0.6,0.8,1.0925,2.27,4.29,5.831,9.6065,13.6090,19.87014,28.599 }; // The cooling sections
float H_Init[Section] = { 1380,1170,980,800,1223.16,735.05,424.32,392.83,328.94,281.64,246.16,160.96 };  // The heat transfer coefficients in the cooling sections
float H_Init_Temp[Section] = { 1380,1170,980,800,1223.16,735.05,424.32,392.83,328.94,281.64,246.16,160.96 };  // The heat transfer coefficients in the cooling sections
float Taim[CoolSection] = { 966.149841, 925.864746, 952.322083, 932.175537, 914.607117, 890.494263, 870.804443, 890.595825 };
float *Mean_TSurface;

void Calculation_MeanTemperature(int nx, int ny, int nz, float dy, float *ccml, float *T);
hipError_t addWithCuda(float *T_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, int tnpts, int num_blocks, int num_threadsx, int num_threadsy);
__device__ void Physicial_Parameters(float T, float *pho, float *Ce, float *lamd);
__device__ float Boundary_Condition(int j, float dx, float *ccml_zone, float *H_Init);

__global__ void addKernel(float *T_New, float *T_Last, float *ccml, float *H_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, bool disout)
{
	int i = threadIdx.x;
	int m = threadIdx.y;
	int j = blockIdx.x;
	int idx = j * nx * nz + m * nx + i;
	int ND = nx * nz;
	int D = nx;

	float pho, Ce, lamd; // physical parameters pho represents desity, Ce is specific heat and lamd is thermal conductivity
	float a, T_Up, T_Down, T_Right, T_Left, T_Forw, T_Back, h = 100.0, Tw = 30.0, Vcast = -0.02, T_Cast = 1558.0;

	if (disout) {
		Physicial_Parameters(T_Last[idx], &pho, &Ce, &lamd);
		a = (lamd) / (pho*Ce);
		h = Boundary_Condition(j, dy, ccml, H_Init);
		if (j == 0) //1
		{
			T_New[idx] = T_Cast;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1)) //10
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = (a*tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ (a*tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m != 0 && m != (nz - 1)) //11
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //12
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //13
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //14
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == 0)  //15
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == (nz - 1))  //16
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == 0)  //17
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == (nz - 1))  //18
		{
			//T_New[idx] = 1550.0;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx - ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //19
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //20
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == 0) //21
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == 0)  //22
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx + D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == (nz - 1)) //23
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == (nz - 1)) //24
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx - 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx - D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m != 0 && m != (nz - 1))  //25
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx + 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //26
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx - 1] - 2 * dx * h * (T_Last[idx] - Tw) / lamd;
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else  //27
		{
			//T_New[idx] = T_Cast;
			T_Up = T_Last[idx + 1];
			T_Down = T_Last[idx - 1];
			T_Right = T_Last[idx + ND];
			T_Left = T_Last[idx - ND];
			T_Forw = T_Last[idx + D];
			T_Back = T_Last[idx - D];
			T_New[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_Last[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}
	}

	else
	{
		Physicial_Parameters(T_New[idx], &pho, &Ce, &lamd);
		a = (lamd) / (pho*Ce);
		h = Boundary_Condition(j, dy, ccml, H_Init);
		if (j == 0) //1
		{
			T_Last[idx] = T_Cast;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m != 0 && m != (nz - 1)) //10
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m != 0 && m != (nz - 1)) //11
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //12
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //13
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //14
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == 0)  //15
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == 0 && m == (nz - 1))  //16
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == 0)  //17
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j == (ny - 1) && i == (nx - 1) && m == (nz - 1))  //18
		{
			//T_Last[idx] = 1550.0;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx - ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == 0)  //19
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i != 0 && i != (nx - 1) && m == (nz - 1))  //20
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D] - 2 * dz * h * (T_Last[idx] - Tw) / lamd;
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == 0) //21
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == 0)  //22
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx + D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m == (nz - 1)) //23
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m == (nz - 1)) //24
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx - 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx - D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == 0 && m != 0 && m != (nz - 1))  //25
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx + 1] - 2 * dx * h * (T_New[idx] - Tw) / lamd;
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else if (j != 0 && j != (ny - 1) && i == (nx - 1) && m != 0 && m != (nz - 1)) //26
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx - 1] - 2 * dx * h * (T_New[idx] - Tw) / lamd;
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}

		else  //27
		{
			//T_Last[idx] = T_Cast;
			T_Up = T_New[idx + 1];
			T_Down = T_New[idx - 1];
			T_Right = T_New[idx + ND];
			T_Left = T_New[idx - ND];
			T_Forw = T_New[idx + D];
			T_Back = T_New[idx - D];
			T_Last[idx] = a*(tao / (dx*dx))*T_Up + a*(tao / (dx*dx))*T_Down + ((1 - 2 * a*tao / (dx*dx) - 2 * a*tao / (dy*dy) - 2 * a*tao / (dz*dz) + tao*Vcast / dy))*T_New[idx]
				+ a*(tao / (dy*dy))*T_Right + (a*tao / (dy*dy) - tao*Vcast / dy)*T_Left + (a*tao / (dz*dz))*T_Forw + (a*tao / (dz*dz))*T_Back;
		}
	}
}

int main()
{
	const int nx = 21, ny = 3000, nz = 21;   // nx is the number of grid in x direction, ny is the number of grid in y direction.
	int num_blocks = 1, num_threadsx = 1, num_threadsy = 1;// num_threadsz = 1; // block number(1D)  thread number in x and y dimension(2D)
	int tnpts = 10001;  // time step
	float T_Cast = 1558.0, Lx = 0.25, Ly = 28.599, Lz = 0.25, t_final = 2000.0, dx, dy, dz, tao;  // T_Cast is the casting temperature Lx and Ly is the thick and length of steel billets
	float *T_Init;

	T_Init = (float *)calloc(nx * ny * nz, sizeof(float));  // Initial condition
	Mean_TSurface = (float*)calloc(Section, sizeof(float));

	num_threadsx = nx;
	num_threadsy = nz;
	num_blocks = ny;

	for (int m = 0; m < nz; m++)
		for (int j = 0; j < ny; j++)
	       for (int i = 0; i < nx; i++)
			   T_Init[nx * ny * m + j * nx + i] = T_Cast;  // give the initial condition

	dx = Lx / (nx - 1);            // the grid size x
	dy = Ly / (ny - 1);            // the grid size y
	dz = Lz / (nz - 1);            // the grid size y
	tao = t_final / (tnpts - 1);   // the time step size
	//gridcheck(dx, dy, tao);

	printf("Casting Temperature = %f ", T_Cast);
	printf("\n");
	printf("The thick of steel billets(m) = %f ", Lx);
	printf("\n");
	printf("The length of steel billets(m) = %f ", Ly);
	printf("\n");
	printf("The length of steel billets(m) = %f ", Lz);
	printf("\n");
	printf("dx(m) = %f ", dx);
	printf("dy(m) = %f ", dy);
	printf("dz(m) = %f ", dz);
	printf("tao(s) = %f ", tao);
	printf("\n");
	printf("simulation time(s) = %f\n ", t_final);

	clock_t timestart = clock();
	hipError_t cudaStatus = addWithCuda(T_Init, dx, dy, dz, tao, nx, ny, nz, tnpts, num_blocks, num_threadsx, num_threadsy);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	clock_t timeend = clock();

	printf("running time = %d(millisecond)", (timeend - timestart));

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

hipError_t addWithCuda(float *T_Init, float dx, float dy, float dz, float tao, int nx, int ny, int nz, int tnpts, int num_blocks, int num_threadsx, int num_threadsy)
{
	hipStream_t stream0, stream1, stream2, stream3, stream4, stream5, stream6, stream7, stream8;
	float *dev_ccml; // the point on GPU
	float *dev_T_New, *dev_T_Last, *dev_H_Init;    // the point on GPU
	float *dev_T_New0, *dev_T_Last0, *dev_H_Init0; // the point on GPU
	float *dev_T_New1, *dev_T_Last1, *dev_H_Init1; // the point on GPU
	float *dev_T_New2, *dev_T_Last2, *dev_H_Init2; // the point on GPU
	float *dev_T_New3, *dev_T_Last3, *dev_H_Init3; // the point on GPU
	float *dev_T_New4, *dev_T_Last4, *dev_H_Init4; // the point on GPU
	float *dev_T_New5, *dev_T_Last5, *dev_H_Init5; // the point on GPU
	float *dev_T_New6, *dev_T_Last6, *dev_H_Init6; // the point on GPU
	float *dev_T_New7, *dev_T_Last7, *dev_H_Init7; // the point on GPU
	float *dev_T_New8, *dev_T_Last8, *dev_H_Init8; // the point on GPU
	float *T_Result, *T_Result0, *T_Result1, *T_Result2, *T_Result3, *T_Result4, *T_Result5, *T_Result6, *T_Result7, *T_Result8;
	float *H_Init_Temp, *H_Init_Temp0, *H_Init_Temp1, *H_Init_Temp2, *H_Init_Temp3, *H_Init_Temp4, *H_Init_Temp5, *H_Init_Temp6, *H_Init_Temp7, *H_Init_Temp8;

	float *Delta_H_Init, *T_HoldLast, **Mean_TSurfaceElement, **Mean_TSurfaceElementOne, **JacobianMatrix;
	float dh = 1.0, arf1, arf2, step = -0.0001;
	const int Num_Iter = 10;      // The result can be obtained by every Num_Iter time step
	const int PrintLabel = 0;     //  If PrintLabel=1 print the information else will not print the information
	volatile bool dstOut = true;
	FILE *fp = NULL;

	Delta_H_Init = (float*)calloc(CoolSection, sizeof(float));

	T_HoldLast = (float*)calloc(nz * ny * nx, sizeof(float));

	JacobianMatrix = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		JacobianMatrix[i] = (float*)calloc(CoolSection, sizeof(float));

	Mean_TSurfaceElement = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		Mean_TSurfaceElement[i] = (float*)calloc(CoolSection, sizeof(float));

	Mean_TSurfaceElementOne = (float**)calloc(CoolSection, sizeof(float));
	for (int i = 0; i < CoolSection; i++)
		Mean_TSurfaceElementOne[i] = (float*)calloc(CoolSection, sizeof(float));

	hipError_t cudaStatus;

	HANDLE_ERROR(hipSetDevice(0));
	HANDLE_ERROR(hipStreamCreate(&stream0));
	HANDLE_ERROR(hipStreamCreate(&stream1));
	HANDLE_ERROR(hipStreamCreate(&stream2));
	HANDLE_ERROR(hipStreamCreate(&stream3));
	HANDLE_ERROR(hipStreamCreate(&stream4));
	HANDLE_ERROR(hipStreamCreate(&stream5));
	HANDLE_ERROR(hipStreamCreate(&stream6));
	HANDLE_ERROR(hipStreamCreate(&stream7));
	HANDLE_ERROR(hipStreamCreate(&stream8));

	// Choose which GPU to run on, change this on a multi-GPU system.
	HANDLE_ERROR(hipSetDevice(0));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result, nx * ny * nz* sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last, nx * ny * nz *  sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init, Section * sizeof(float)));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result0, nx * ny * nz * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp0, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New0, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last0, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init0, Section * sizeof(float)));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result1, nx * ny * nz * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp1, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New1, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last1, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init1, Section * sizeof(float)));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result2, nx * ny * nz * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp2, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New2, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last2, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init2, Section * sizeof(float)));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result3, nx * ny * nz * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp3, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New3, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last3, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init3, Section * sizeof(float)));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result4, nx * ny * nz * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp4, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New4, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last4, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init4, Section * sizeof(float)));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result5, nx * ny * nz * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp5, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New5, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last5, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init5, Section * sizeof(float)));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result6, nx * ny * nz * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp6, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New6, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last6, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init6, Section * sizeof(float)));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result7, nx * ny * nz * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp7, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New7, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last7, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init7, Section * sizeof(float)));

	HANDLE_ERROR(hipHostAlloc((void**)&T_Result8, nx * ny * nz * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&H_Init_Temp8, Section * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_New8, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_T_Last8, nx * ny * nz * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_H_Init8, Section * sizeof(float)));

	HANDLE_ERROR(hipMalloc((void**)&dev_ccml, (Section + 1) * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_T_Last, T_Init, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_ccml, ccml, (Section + 1) * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init, Section * sizeof(float), hipMemcpyHostToDevice));

	dim3 threadsPerBlock(num_threadsx, num_threadsy);

	for (int i = 0; i < tnpts; i++)
	{
		if (i % Num_Iter == 0)
		{
			for (int temp = 0; temp < Section; temp++)
				H_Init_Temp0[temp] = H_Init[temp];
			for (int temp = 0; temp < Section; temp++)
				H_Init_Temp1[temp] = H_Init[temp];
			H_Init_Temp1[0 + MoldSection] = H_Init[0 + MoldSection] + dh;
			for (int temp = 0; temp < Section; temp++)
				H_Init_Temp2[temp] = H_Init[temp];
			H_Init_Temp2[1 + MoldSection] = H_Init[1 + MoldSection] + dh;
			for (int temp = 0; temp < Section; temp++)
				H_Init_Temp3[temp] = H_Init[temp];
			H_Init_Temp3[2 + MoldSection] = H_Init[2 + MoldSection] + dh;
			for (int temp = 0; temp < Section; temp++)
				H_Init_Temp4[temp] = H_Init[temp];
			H_Init_Temp4[3 + MoldSection] = H_Init[3 + MoldSection] + dh;
			for (int temp = 0; temp < Section; temp++)
				H_Init_Temp5[temp] = H_Init[temp];
			H_Init_Temp5[4 + MoldSection] = H_Init[4 + MoldSection] + dh;
			for (int temp = 0; temp < Section; temp++)
				H_Init_Temp6[temp] = H_Init[temp];
			H_Init_Temp6[5 + MoldSection] = H_Init[5 + MoldSection] + dh;
			for (int temp = 0; temp < Section; temp++)
				H_Init_Temp7[temp] = H_Init[temp];
			H_Init_Temp7[6 + MoldSection] = H_Init[6 + MoldSection] + dh;
			for (int temp = 0; temp < Section; temp++)
				H_Init_Temp8[temp] = H_Init[temp];
			H_Init_Temp8[7 + MoldSection] = H_Init[7 + MoldSection] + dh;

			HANDLE_ERROR(hipMemcpy(T_HoldLast, dev_T_Last, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));

			//  Section 0
			HANDLE_ERROR(hipMemcpyAsync(dev_H_Init0, H_Init_Temp0, Section * sizeof(float), hipMemcpyHostToDevice, stream0));
			HANDLE_ERROR(hipMemcpyAsync(dev_T_Last0, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice, stream0));
			for (int PNum = 0; PNum < Num_Iter; PNum++)
			{
				addKernel << <num_blocks, threadsPerBlock, 0, stream0 >> > (dev_T_New0, dev_T_Last0, dev_ccml, dev_H_Init0, dx, dy, dz, tao, nx, ny, nz, dstOut);
				dstOut = !dstOut;
			}
			HANDLE_ERROR(hipMemcpyAsync(T_Result0, dev_T_Last0, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost, stream0));

			//Section 1
			HANDLE_ERROR(hipMemcpyAsync(dev_H_Init1, H_Init_Temp1, Section * sizeof(float), hipMemcpyHostToDevice, stream1));
			HANDLE_ERROR(hipMemcpyAsync(dev_T_Last1, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice, stream1));
			for (int PNum = 0; PNum < Num_Iter; PNum++)
			{
				addKernel << <num_blocks, threadsPerBlock, 0, stream1 >> > (dev_T_New1, dev_T_Last1, dev_ccml, dev_H_Init1, dx, dy, dz, tao, nx, ny, nz, dstOut);
				dstOut = !dstOut;
			}
			HANDLE_ERROR(hipMemcpyAsync(T_Result1, dev_T_Last1, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost, stream1));


			//Section 2
			HANDLE_ERROR(hipMemcpyAsync(dev_H_Init2, H_Init_Temp2, Section * sizeof(float), hipMemcpyHostToDevice, stream2));
			HANDLE_ERROR(hipMemcpyAsync(dev_T_Last2, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice, stream2));
			for (int PNum = 0; PNum < Num_Iter; PNum++)
			{
				addKernel << <num_blocks, threadsPerBlock, 0, stream2 >> > (dev_T_New2, dev_T_Last2, dev_ccml, dev_H_Init2, dx, dy, dz, tao, nx, ny, nz, dstOut);
				dstOut = !dstOut;
			}
			HANDLE_ERROR(hipMemcpyAsync(T_Result2, dev_T_Last2, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost, stream2));

			//Section 3
			HANDLE_ERROR(hipMemcpyAsync(dev_H_Init3, H_Init_Temp3, Section * sizeof(float), hipMemcpyHostToDevice, stream3));
			HANDLE_ERROR(hipMemcpyAsync(dev_T_Last3, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice, stream3));
			for (int PNum = 0; PNum < Num_Iter; PNum++)
			{
				addKernel << <num_blocks, threadsPerBlock, 0, stream3 >> > (dev_T_New3, dev_T_Last3, dev_ccml, dev_H_Init3, dx, dy, dz, tao, nx, ny, nz, dstOut);
				dstOut = !dstOut;
			}
			HANDLE_ERROR(hipMemcpyAsync(T_Result3, dev_T_Last3, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost, stream3));

			//Section 4
			HANDLE_ERROR(hipMemcpyAsync(dev_H_Init4, H_Init_Temp4, Section * sizeof(float), hipMemcpyHostToDevice, stream4));
			HANDLE_ERROR(hipMemcpyAsync(dev_T_Last4, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice, stream4));
			for (int PNum = 0; PNum < Num_Iter; PNum++)
			{
				addKernel << <num_blocks, threadsPerBlock, 0, stream4 >> > (dev_T_New4, dev_T_Last4, dev_ccml, dev_H_Init4, dx, dy, dz, tao, nx, ny, nz, dstOut);
				dstOut = !dstOut;
			}
			HANDLE_ERROR(hipMemcpyAsync(T_Result4, dev_T_Last4, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost, stream4));

			//Section 5
			HANDLE_ERROR(hipMemcpyAsync(dev_H_Init5, H_Init_Temp5, Section * sizeof(float), hipMemcpyHostToDevice, stream5));
			HANDLE_ERROR(hipMemcpyAsync(dev_T_Last5, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice, stream5));
			for (int PNum = 0; PNum < Num_Iter; PNum++)
			{
				addKernel << <num_blocks, threadsPerBlock, 0, stream5 >> > (dev_T_New5, dev_T_Last5, dev_ccml, dev_H_Init5, dx, dy, dz, tao, nx, ny, nz, dstOut);
				dstOut = !dstOut;
			}
			HANDLE_ERROR(hipMemcpyAsync(T_Result5, dev_T_Last5, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost, stream5));


			//Section 6
			HANDLE_ERROR(hipMemcpyAsync(dev_H_Init6, H_Init_Temp6, Section * sizeof(float), hipMemcpyHostToDevice, stream6));
			HANDLE_ERROR(hipMemcpyAsync(dev_T_Last6, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice, stream6));
			for (int PNum = 0; PNum < Num_Iter; PNum++)
			{
				addKernel << <num_blocks, threadsPerBlock, 0, stream6 >> > (dev_T_New6, dev_T_Last6, dev_ccml, dev_H_Init6, dx, dy, dz, tao, nx, ny, nz, dstOut);
				dstOut = !dstOut;
			}
			HANDLE_ERROR(hipMemcpyAsync(T_Result6, dev_T_Last6, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost, stream6));


			//Section 7
			HANDLE_ERROR(hipMemcpyAsync(dev_H_Init7, H_Init_Temp7, Section * sizeof(float), hipMemcpyHostToDevice, stream7));
			HANDLE_ERROR(hipMemcpyAsync(dev_T_Last7, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice, stream7));
			for (int PNum = 0; PNum < Num_Iter; PNum++)
			{
				addKernel << <num_blocks, threadsPerBlock, 0, stream7 >> > (dev_T_New7, dev_T_Last7, dev_ccml, dev_H_Init7, dx, dy, dz, tao, nx, ny, nz, dstOut);
				dstOut = !dstOut;
			}
			HANDLE_ERROR(hipMemcpyAsync(T_Result7, dev_T_Last7, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost, stream7));


			//Section 8
			HANDLE_ERROR(hipMemcpyAsync(dev_H_Init8, H_Init_Temp8, Section * sizeof(float), hipMemcpyHostToDevice, stream8));
			HANDLE_ERROR(hipMemcpyAsync(dev_T_Last8, T_HoldLast, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice, stream8));
			for (int PNum = 0; PNum < Num_Iter; PNum++)
			{
				addKernel << <num_blocks, threadsPerBlock, 0, stream8 >> > (dev_T_New8, dev_T_Last8, dev_ccml, dev_H_Init8, dx, dy, dz, tao, nx, ny, nz, dstOut);
				dstOut = !dstOut;
			}
			HANDLE_ERROR(hipMemcpyAsync(T_Result8, dev_T_Last8, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost, stream8));

			// stream synchronize
			HANDLE_ERROR(hipStreamSynchronize(stream0));
			HANDLE_ERROR(hipStreamSynchronize(stream1));
			HANDLE_ERROR(hipStreamSynchronize(stream2));
			HANDLE_ERROR(hipStreamSynchronize(stream3));
			HANDLE_ERROR(hipStreamSynchronize(stream4));
			HANDLE_ERROR(hipStreamSynchronize(stream5));
			HANDLE_ERROR(hipStreamSynchronize(stream6));
			HANDLE_ERROR(hipStreamSynchronize(stream7));
			HANDLE_ERROR(hipStreamSynchronize(stream8));


			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result0);  // calculation the mean surface temperature of steel billets in every cooling sections
			for (int temp = 0; temp < CoolSection; temp++)
			{
				for (int column = 0; column < CoolSection; column++)
					Mean_TSurfaceElementOne[temp][column] = Mean_TSurface[column + MoldSection];
				//Mean_TSurfaceElementOne[CoolSection - 1][CoolSection - 1] = Mean_TSurface[column + MoldSection];
			}

			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result1);   // calculation the mean surface temperature of steel billets in every cooling sections
			for (int column = 0; column < CoolSection; column++)
				Mean_TSurfaceElement[0][column] = Mean_TSurface[column + MoldSection];

			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result2);   // calculation the mean surface temperature of steel billets in every cooling sections
			for (int column = 0; column < CoolSection; column++)
				Mean_TSurfaceElement[1][column] = Mean_TSurface[column + MoldSection];

			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result3);   // calculation the mean surface temperature of steel billets in every cooling sections
			for (int column = 0; column < CoolSection; column++)
				Mean_TSurfaceElement[2][column] = Mean_TSurface[column + MoldSection];

			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result4);  // calculation the mean surface temperature of steel billets in every cooling sections
			for (int column = 0; column < CoolSection; column++)
				Mean_TSurfaceElement[3][column] = Mean_TSurface[column + MoldSection];

			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result5);   // calculation the mean surface temperature of steel billets in every cooling sections
			for (int column = 0; column < CoolSection; column++)
				Mean_TSurfaceElement[4][column] = Mean_TSurface[column + MoldSection];

			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result6);   // calculation the mean surface temperature of steel billets in every cooling sections
			for (int column = 0; column < CoolSection; column++)
				Mean_TSurfaceElement[5][column] = Mean_TSurface[column + MoldSection];

			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result7);   // calculation the mean surface temperature of steel billets in every cooling sections
			for (int column = 0; column < CoolSection; column++)
				Mean_TSurfaceElement[6][column] = Mean_TSurface[column + MoldSection];

			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result8);   // calculation the mean surface temperature of steel billets in every cooling sections
			for (int column = 0; column < CoolSection; column++)
				Mean_TSurfaceElement[7][column] = Mean_TSurface[column + MoldSection];


			if (PrintLabel == 1)
				printf("\nJacobianMatrix=\n");
			for (int row = 0; row < CoolSection; row++)
			{
				for (int column = 0; column < CoolSection; column++)
				{
					JacobianMatrix[row][column] = (Mean_TSurfaceElement[row][column] - Mean_TSurfaceElementOne[row][column]) / dh;
					if (PrintLabel == 1)
						printf("%f ", JacobianMatrix[row][column]);
				}
				if (PrintLabel == 1)
					printf("\n");
			}

			for (int temp = 0; temp < CoolSection; temp++)
				Delta_H_Init[temp] = 0.0;

			if (PrintLabel == 1)
				printf("\nDelta_H_Init=\n");
			for (int temp = 0; temp < CoolSection; temp++)
			{
				for (int column = 0; column < CoolSection; column++)
					Delta_H_Init[temp] += (Mean_TSurfaceElementOne[temp][column] - Taim[column]) * JacobianMatrix[temp][column];
				if (PrintLabel == 1)
					printf(" %f, ", Delta_H_Init[temp]);
			}

			arf1 = 0.0, arf2 = 0.0;
			for (int temp = 0; temp < CoolSection; temp++)
			{
				for (int column = 0; column < CoolSection; column++)
				{
					arf1 += (Mean_TSurfaceElementOne[0][temp] - Taim[temp]) * JacobianMatrix[temp][column] * Delta_H_Init[column];
					arf2 += JacobianMatrix[temp][column] * Delta_H_Init[column] * JacobianMatrix[temp][column] * Delta_H_Init[column];
				}
			}
			step = -arf1 / (arf2);

			for (int temp = 0; temp < CoolSection; temp++)
				H_Init[temp + MoldSection] += step *(Delta_H_Init[temp]);
		}

		for (int temp = 0; temp < Section; temp++)
			H_Init_Temp[temp] = H_Init[temp];
		HANDLE_ERROR(hipMemcpy(dev_H_Init, H_Init_Temp, Section * sizeof(float), hipMemcpyHostToDevice));
		addKernel << <num_blocks, threadsPerBlock >> > (dev_T_New, dev_T_Last, dev_ccml, dev_H_Init, dx, dy, dz, tao, nx, ny, nz, dstOut);
		dstOut = !dstOut;

		if (i % Num_Iter == 0)
		{
			HANDLE_ERROR(hipMemcpy(T_Result, dev_T_Last, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));

			Calculation_MeanTemperature(nx, ny, nz, dy, ccml, T_Result); ;  // calculation the mean surface temperature of steel billets in every cooling sections

			if (PrintLabel == 1)
			{
				printf("time_step = %d  simulation time is %f\n", i, i*tao);
				printf("\nTSurface=\n");
				for (int temp = 0; temp < CoolSection; temp++)
					printf("%f, ", Mean_TSurface[temp + MoldSection]);

				printf("\nTSurface - Taim=\n");
				for (int temp = 0; temp < CoolSection; temp++)
					printf("%f, ", Mean_TSurface[temp + MoldSection] - Taim[temp]);
			}
		}
	}

	/*fp = fopen("D:\\Temperature3DGPU_Static.txt", "w");
	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			for (int m = 0; m < nz; m++)
				fprintf(fp, " %f", T_Result[nx * nz * j  + i * nz + m]);
			fprintf(fp, "\n");
		}
		fprintf(fp, "\n");
	}
	fclose(fp);

	fp = fopen("D:\\SurfaceTemperature3DGPU_Static.txt", "w");
	for (int j = 0; j < ny; j++)
	{
		fprintf(fp, "%f", T_Result[nx * nz * j + 0 * nz + int((nx - 1) / 2)]);
		fprintf(fp, "\n");
	}
	fclose(fp);*/


	

	// Check for any errors launching the kernel
	HANDLE_ERROR(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.


Error:
	hipFree(dev_T_New);
	hipFree(dev_T_Last);
	hipFree(dev_ccml);
	hipFree(dev_H_Init);

	return cudaStatus;
}
// Helper function for using CUDA to add vectors in parallel.

__device__ void Physicial_Parameters(float T, float *pho, float *Ce, float *lamd)
{
	float Ts = 1462.0, Tl = 1518.0, lamds = 30, lamdl = 50, phos = 7000, phol = 7500, ce = 540.0, L = 265600.0, fs = 0.0;
	if (T<Ts)
	{
		fs = 0;
		*pho = phos;
		*lamd = lamds;
		*Ce = ce;
	}

	if (T >= Ts&&T <= Tl)
	{
		fs = (T - Ts) / (Tl - Ts);
		*pho = fs*phos + (1 - fs)*phol;
		*lamd = fs*lamds + (1 - fs)*lamdl;
		*Ce = ce + L / (Tl - Ts);
	}

	if (T>Tl)
	{
		fs = 1;
		*pho = phol;
		*lamd = lamdl;
		*Ce = ce;
	}

}

__device__ float Boundary_Condition(int j, float dy, float *ccml_zone, float *H_Init)
{
	float YLabel, h = 0.0;
	YLabel = j*dy;

	for (int i = 0; i < Section; i++)
	{
		if (YLabel >= *(ccml_zone + i) && YLabel <= *(ccml_zone + i + 1))
			h = *(H_Init + i);
	}
	return h;
}

void Calculation_MeanTemperature(int nx, int ny, int nz, float dy, float *ccml, float *T)
{
	float y;
	int count = 0;
	int i = 0;
	
	for (int i = 0; i < Section; i++)
	{
		Mean_TSurface[i] = 0.0;
		for (int j = 0; j < ny; j++)
		{
			y = j * dy;
			if (y > *(ccml + i) && y <= *(ccml + i + 1))
			{
				Mean_TSurface[i] = Mean_TSurface[i] + T[nx * nz * j + 0 * nz + int((nx - 1) / 2)];
				count++;
			}
		}
		Mean_TSurface[i] = Mean_TSurface[i] / float(count);
		count = 0;
	}
}